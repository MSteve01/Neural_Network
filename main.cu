#include "hip/hip_runtime.h"
#include "Header.cuh"
#include "Neural_Network.cu"
#include <thread>
#include <chrono>

// file name
const char* Model_file_name = "file/Model.txt";
const char* DataBase_file_name = "file/Data.txt";
const char* output_file_name = "file/Output.txt";
const char* RandWeightSetting_file_name = "file/Rand_weight_setting.txt";
const char* RandBiasSetting_file_name = "file/Rand_bias_setting.txt";
const char* Lost_file_name = "file/lost.txt";

// variable
int data_range;
int learning_range;
int testing_range;
int input_range;
int output_range;
int have_trained = 0;
double train_speed = 1;
int load_range = 0;

// weight and bias initialization function
std::function<double()>
random_func = []() {
	//return std::pow(-1, rand() % 2) * double(rand() % 20000) / 20000;
	return std::pow(-1, rand() % 2) * std::tanh(double(rand() % 30000) / 10000);
};
std::function<double(std::size_t, std::size_t)> 
random_func2 = [](std::size_t size, std::size_t next) {
	return std::pow(-1, rand() % 2) * (double(rand() % 20000) / 10000) * std::sqrt(double(2) / size);
};
std::function<double()> 
zero = []() {
	return 0;
};


std::vector<LayerId> load_model() {
	std::vector<LayerId> Model;
	std::ifstream Model_file(Model_file_name);
	while (!Model_file.eof()) {
		int input1, input2; Model_file >> input1 >> input2;
		std::string setting; std::getline(Model_file, setting);
		Model.push_back(LayerId(Layer::type(input1), input2, setting));
	}
	Model_file.close();
	return Model;
}

std::vector<Matrix<double>> load_data(std::size_t input_size) { // loas the whole data
	std::vector<Matrix<double>> Data;
	std::ifstream DataBase_file(DataBase_file_name);
	while (!DataBase_file.eof()) {
		Data.push_back(Matrix<double>(input_size,1));
		double* get_input = new double[input_size];
		for (int i = 0; i < input_size; i++) {
			DataBase_file >> get_input[i];
		}
		hipMemcpy(Data.back().get_value(), get_input, input_size * sizeof(double), hipMemcpyHostToDevice);
		delete[] get_input;
	}
	return Data;
}

std::vector<Matrix<double>> load_data(std::size_t input_size, std::size_t data_range) { // load the data in specific range
	int loop = 0;
	std::vector<Matrix<double>> Data;
	static std::ifstream DataBase_file(DataBase_file_name);
	while (!DataBase_file.eof() && loop < data_range) {
		Data.push_back(Matrix<double>(input_size, 1));
		double* get_input = new double[input_size];
		for (int i = 0; i < input_size; i++) {
			DataBase_file >> get_input[i];
		}
		hipMemcpy(Data.back().get_value(), get_input, input_size * sizeof(double), hipMemcpyHostToDevice);
		delete[] get_input;
		loop++;
	}
	return Data;
}

std::vector<std::pair<double, double>> load_rand_weight_setting() {
	std::vector<std::pair<double, double>> setting;
	std::ifstream RandWeightFile(RandWeightSetting_file_name);
	while (!RandWeightFile.eof()) {
		double input1, input2;
		RandWeightFile >> input1 >> input2;
		setting.push_back({ input1, input2 });
	}
	RandWeightFile.close();
	return setting;
}

std::vector<std::pair<double, double>> load_rand_bias_setting() {
	std::vector<std::pair<double, double>> setting;
	std::ifstream RandBiasFile(RandBiasSetting_file_name);
	while (!RandBiasFile.eof()) {
		double input1, input2;
		RandBiasFile >> input1 >> input2;
		setting.push_back({ input1,input2 });
	}
	RandBiasFile.close();
	return setting;
}



double learn(Neural_Network& AI, std::vector<Matrix<double>> Data, int start) {
	double lost = 0;
	for (int i = start; i < start + input_range; i++) {															// feedforward
		AI.feedforward(Data[i]);
	}

	AI.mul_change_dependencies(0);
	for (int i = start + input_range; i < start + input_range + output_range; i++) {							// backpropagation
		AI.backpropagation(Data[i]);
		lost += AI.get_loss(Data[i]);
	}

	have_trained++;
	AI.change_dependencies();
	AI.fogot_all();

	return lost;
}

std::vector<Matrix<double>> predict(Neural_Network& AI, std::vector<Matrix<double>> Data, int start) {
	std::vector<Matrix<double>> result;
	for (int i = start; i < start + input_range; i++) {
		AI.feedforward(Data[i]);
	}

	for (int i = start + input_range; i < start + input_range + output_range; i++) {
		result.push_back(Matrix<double>(AI.get_input_size(), 1)); result.back() = AI.get_output();
		//AI.feedforward(result.back());
	}

	AI.fogot_all();
	return result;
}

char get_char(const Matrix<double>& M) {
	int max = -1000000;
	int pos = 0;
	double* a = new double[M.get_size()];
	hipMemcpy(a, M.get_value(), M.get_sizeb(), hipMemcpyDeviceToHost);
	for(int i = 0 ;i<M.get_size();i++) {
		if (max < a[i]) {
			max = a[i];
			pos = i;
		}
	}
	delete[] a;
	return pos;
}

int main() {
	try {
		std::srand(std::time(0));
		Neural_Network AI(load_model(), catagorical_CEnt_loss_func, dcatagorical_CEnt_loss_func); 
		std::cout << "Model was leaded successfully\n";
		
		std::vector<Matrix<double>> Data;
		std::cout << "data range : "; std::cin >> data_range;													// get setting 
		std::cout << "load_range : "; std::cin >> load_range; 
		std::cout << "learing range : ";std::cin >> learning_range;	
		std::cout << "input range : "; std::cin >> input_range;
		std::cout << "output range : "; std::cin >> output_range;
		
		testing_range = data_range - learning_range;
		
		std::vector<std::function<double(std::size_t, std::size_t)>> Weight_setting; 
		for (int i = 0; i < AI.get_layer_size() - 1; i++) { Weight_setting.push_back(random_func2); }
		std::cout << "Load rand weigth setting successfully\n";
		std::vector<std::function<double()>> Bias_setting;
		for (int i = 0; i < AI.get_layer_size() - 1; i++) { Bias_setting.push_back(zero); }
		std::cout << "Load rand bias setting successfully\n";

		if (Weight_setting.size() < AI.get_layer_size() - 1) {													// check for error setting
			std::cout << "Weight setting doesn't match AIsize\n"; return 0;
		}
		if (Bias_setting.size() < AI.get_layer_size() - 1) {
			std::cout << "Bias setting doesn't maych AIsize\n"; return 0;
		}



		AI.set_all_learning_rate(0.001);																		// set up AI
		AI.rand_weight(Weight_setting);
		AI.rand_bias(Bias_setting);
		AI.set_change_dependencies(0);

		std::ofstream output_file(output_file_name);
		std::ofstream lost_file(Lost_file_name);

		for (int i = 0; i + input_range + output_range < learning_range; i++) {									// loop though every data for learning
			if (i >= 1000000)
				AI.set_all_learning_rate(0.0005);
			else if (i >= 200000)
				AI.set_all_learning_rate(0.00001);
			if (i % load_range == 0) {																			// loas data
				Data = load_data(AI.get_input_size(), load_range);
			}

			int pos = rand() % ( load_range - input_range - output_range);										// random pattern for training

			lost_file << learn(AI, Data, pos) << "\n";															// learn and put lost into the file

			std::cout << get_char(AI.get_output()) << "\t|\t" << get_char(Data[i + input_range + output_range]) << std::endl;
		}
		std::cout << "started testing\n";																		// predict
		Data = load_data(AI.get_input_size(), input_range);
		for (int i = 0; i + input_range + output_range < testing_range; i+=output_range) {
			std::vector<Matrix<double>> output = predict(AI, Data, i);
			for (int k = 0; k < output_range; k++) {
				char g = get_char(output[k]);
				output_file << g << std::endl;
				Data.push_back(output[k]);
			}
		}
		//	s1.join();
		return 0;
	}
	catch (std::string Error) {																					// catch the error
		std::cout << Error << std::endl;
		std::cin.get();
		return 0;
	}
	std::cin.get();
}